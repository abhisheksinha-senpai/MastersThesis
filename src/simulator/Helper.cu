#include "hip/hip_runtime.h"
#include "Helper.cuh"

unsigned int SCR_WIDTH = 800;
unsigned int SCR_HEIGHT = 600;
float deltaTime = 0.0f;
float lastFrame = 0.0f;
float camradius = 7.0f;
float cameraspeed = 0.02f;
float camX = camradius;
float camY = 0.0f;
float camZ = 0.0f;
bool firstMouse = true;
float yaw = -90.0f;
float pitch = 0.0f;
float lastX = SCR_WIDTH / 2.0;
float lastY = SCR_HEIGHT / 2.0;
float fov = 45.0f;

glm::vec3 cameraPos = glm::vec3(camX, camY, camZ);
glm::vec3 cameraFront = glm::vec3(0.0f, 0.0f, -1.0f);
glm::vec3 cameraUp = glm::vec3(0.0f, 1.0f, 0.0f);

glm::mat4 view = glm::mat4(1.0f);
glm::mat4 model = glm::mat4(1.0f);
glm::mat4 proj = glm::mat4(1.0f);


__host__ void mouse_callback(GLFWwindow* window, double xposIn, double yposIn);
__host__ void framebuffer_size_callback(GLFWwindow* window, int width, int height);

__host__ void display_init(GLFWwindow** window)
{
    glfwInit();
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    (*window) = glfwCreateWindow(SCR_WIDTH, SCR_HEIGHT, "LearnOpenGL", NULL, NULL);

    if (*window == NULL)
    {
        std::cout << "Failed to create GLFW window" << std::endl;
        glfwTerminate();
        return;
    }
    glfwMakeContextCurrent(*window);
    glfwSetFramebufferSizeCallback(*window, framebuffer_size_callback);
    glfwSetCursorPosCallback(*window, mouse_callback);
    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
    {
        std::cout << "Failed to initialize GLAD" << std::endl;
        return;
    }
    stbi_set_flip_vertically_on_load(true);
    //glEnable(GL_DEPTH_TEST);
    glfwSetInputMode(*window, GLFW_CURSOR, GLFW_CURSOR_NORMAL );
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
    glEnable( GL_BLEND );
    printf("%s %s %s\n", glGetString(GL_VERSION), glGetString(GL_VENDOR), glGetString(GL_RENDERER));
    printf("Display initialized.....\n");
}

__host__ void model_init(ResourceManager &r_manager, Shader &ourShader, Model &ourModel, int NX, int NY, int NZ,
glm::f32vec3 scale, glm::f32vec3 origin)
{
    r_manager.load_shader("resources/shaders/vertex/model_shader.vs", "VERTEX", ourShader.vertex_shader);
    r_manager.load_shader("resources/shaders/fragment/model_shader.fs", "FRAGMENT", ourShader.fragment_shader);
    ourShader.create_vs_shader(ourShader.vertex_shader.c_str());
    ourShader.create_fs_shader(ourShader.fragment_shader.c_str());
    ourShader.compile();
    // std::string model_name = "resources/backpack/backpack.obj";
    // std::string model_name = "resources/Cup/cup.obj";
    // std::string model_name = "resources/Cube/slime.obj";
    std::string model_name = "resources/Sphere/Ball.obj";
    ourModel = Model((char *)model_name.c_str(), scale, origin);
    printf("Model initialized.....\n");
}

void domain_init(int NX, int NY, int NZ,
                float **rho, float **ux, float **uy,float **uz)
{
    int sz = NX*NY*NZ*sizeof(float);
    *rho = (float *)malloc(sz);
    *ux =  (float *)malloc(sz);
    *uy =  (float *)malloc(sz);
    *uz =  (float *)malloc(sz);

    memset(*rho, 0, sz);
    memset(*ux, 0, sz);
    memset(*uy, 0, sz);
    memset(*uz, 0, sz);
    
    int loc = 0, X1, Y1, Z1;
    for(int j=0;j<NY;j++)
    {
        for(int i=0;i<NX;i++)
        {
            for(int k=0;k<NZ;k++)
            {
                loc = i+j*NX+k*NX*NY;
                (*rho)[loc] = 1.0f;
                (*ux)[loc] = 0.0f;
                (*uy)[loc] = 0.0f;
                (*uz)[loc] = 0.0f;
            }
        }
    }

    // for(int i =0;i<obj.meshes.size();i++)
    // {
    //     for(int j=0;j<obj.meshes[i].size();j++)
    //     {
    //         X1 = (int)obj.meshes[i][j].Position.x;
    //         Y1 = (int)obj.meshes[i][j].Position.y;
    //         Z1 = (int)obj.meshes[i][j].Position.z;
    //         loc  = i+j*NX+k*NX*NY;
    //         (*ux)[loc] = obj.meshes[i][j].Base_Vel.x;
    //         (*uy)[loc] = obj.meshes[i][j].Base_Vel.z;
    //         (*uz)[loc] = obj.meshes[i][j].Base_Vel.z;

    //     }
    // }
    printf("Domain initialized...\n");
}

__host__ void scene_init(float *rho_gpu, float *ux_gpu, float *uy_gpu, float *uz_gpu,
                         float *rho, float *ux, float *uy, float *uz, 
                         int NX, int NY, int NZ)
{
    int sz = NX*NY*NZ*sizeof(float);
    checkCudaErrors(hipMemcpy(rho_gpu, rho, sz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(ux_gpu, ux, sz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(uy_gpu, uy, sz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(uz_gpu, uz, sz, hipMemcpyHostToDevice));
    printf("Scene initialized.....\n");
}

__host__ void scene_cleanup(Vertex **nodeLists, Vertex **nodeData, int *vertex_size_per_mesh,
                            float *rho, float *ux, float *uy, float *uz)
{
    free(rho);
    free(ux);
    free(uy);
    free(uz);
    free(nodeLists);
    free(nodeData);
    free(vertex_size_per_mesh);
}

__host__ void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
    glViewport(0, 0, width, height);
}

__host__ void processInput(GLFWwindow* window)
{
    float currentFrame = glfwGetTime();
    deltaTime = currentFrame - lastFrame;
    lastFrame = currentFrame;

    if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
        glfwSetWindowShouldClose(window, true);

    const float cameraSpeed = 5.0f * deltaTime; // adjust accordingly
    if (glfwGetKey(window, GLFW_KEY_W) == GLFW_PRESS)
        cameraPos += cameraSpeed * cameraFront;
    if (glfwGetKey(window, GLFW_KEY_S) == GLFW_PRESS)
        cameraPos -= cameraSpeed * cameraFront;
    if (glfwGetKey(window, GLFW_KEY_A) == GLFW_PRESS)
        cameraPos -= glm::normalize(glm::cross(cameraFront, cameraUp)) * cameraSpeed;
    if (glfwGetKey(window, GLFW_KEY_D) == GLFW_PRESS)
        cameraPos += glm::normalize(glm::cross(cameraFront, cameraUp)) * cameraSpeed;
}

__host__ void mouse_callback(GLFWwindow* window, double xposIn, double yposIn)
{
    if (glfwGetMouseButton(window, GLFW_MOUSE_BUTTON_RIGHT) == GLFW_PRESS)
    {
        float xpos = static_cast<float>(xposIn);
        float ypos = static_cast<float>(yposIn);

        if (firstMouse)
        {
            lastX = xpos;
            lastY = ypos;
            firstMouse = false;
        }

        float xoffset = xpos - lastX;
        float yoffset = lastY - ypos; // reversed since y-coordinates go from bottom to top
        lastX = xpos;
        lastY = ypos;

        float sensitivity = 0.25f; // change this value to your liking
        xoffset *= sensitivity;
        yoffset *= sensitivity;

        yaw += xoffset;
        pitch += yoffset;

        // make sure that when pitch is out of bounds, screen doesn't get flipped
        if (pitch > 89.0f)
            pitch = 89.0f;
        if (pitch < -89.0f)
            pitch = -89.0f;

        glm::vec3 front;
        front.x = cos(glm::radians(yaw)) * cos(glm::radians(pitch));
        front.y = sin(glm::radians(pitch));
        front.z = sin(glm::radians(yaw)) * cos(glm::radians(pitch));

        cameraFront = glm::normalize(front);
    }
    else if (glfwGetMouseButton(window, GLFW_MOUSE_BUTTON_RIGHT) == GLFW_RELEASE)
    {
        lastX = SCR_WIDTH / 2.0;
        lastY = SCR_HEIGHT / 2.0;
        firstMouse = true;
    }
}

__host__ void transfer_mesh_data(int num_mesh, Vertex **nodeLists, 
                                int *vertex_size_per_mesh,
                                hipStream_t *streams, Model &objModel)
{
    for(int i=0;i<num_mesh;i++)
        checkCudaErrors(hipMemcpyAsync(&objModel.meshes[i].vertices[0], nodeLists[i], vertex_size_per_mesh[i]*sizeof(Vertex), hipMemcpyDeviceToHost, streams[i]));
}

__host__ void draw_model( GLFWwindow* window, Shader& shader, Model& objmodel,
                          glm::f32vec3 scale, 
                          int num_mesh, Vertex **nodeLists, 
                          int *vertex_size_per_mesh,
                          hipStream_t *streams)
{
    transfer_mesh_data(num_mesh, nodeLists, vertex_size_per_mesh, streams, objmodel);
    checkCudaErrors(hipDeviceSynchronize());
    shader.use();
    // view/projection transformations
    view = glm::lookAt(cameraPos, cameraPos+cameraFront, cameraUp);
    proj = glm::perspective(glm::radians(45.0f), (float)SCR_WIDTH / (float)SCR_HEIGHT, 0.1f, 1000.0f);

    glUniformMatrix4fv(glGetUniformLocation(shader.get_shader_pgm(), "view"), 1, GL_FALSE, glm::value_ptr(view));
    glUniformMatrix4fv(glGetUniformLocation(shader.get_shader_pgm(), "projection"), 1, GL_FALSE, glm::value_ptr(proj));
    // render the loaded model
    objmodel.Draw(shader, scale);
    model = glm::mat4(1);
}

__host__ void transfer_fluid_data(float *rho, float*ux, float *uy,float *uz,
                                  float *rho_gpu, float *ux_gpu, float*uy_gpu, float* uz_gpu, 
                                  int NX, int NY, int NZ)
{
    int sz = NX*NY*NZ*sizeof(float);
    hipMemcpy(rho, (void *)rho_gpu, sz, hipMemcpyDeviceToHost);
    hipMemcpy(ux, (void *)ux_gpu, sz, hipMemcpyDeviceToHost);
    hipMemcpy(uy, (void *)uy_gpu, sz, hipMemcpyDeviceToHost);
    hipMemcpy(uz, (void *)uz_gpu, sz, hipMemcpyDeviceToHost);
}

__host__ void draw_fluid(float *rho, float*ux, float *uy, float *uz,
                         float *rho_gpu, float *ux_gpu, float*uy_gpu, float* uz_gpu,
                         int NX, int NY, int NZ, 
                         ParticleSystem &fluid, glm::f32vec3 model_scale, glm::f32vec3 dis_scale)
{
    transfer_fluid_data(rho, ux, uy, uz,
                        rho_gpu, ux_gpu, uy_gpu, uz_gpu,
                        NX, NY, NZ);
    
    fluid.update_particles(NX, NY, NZ, rho, ux, uy, uz, model_scale);
    fluid.draw_particles(SCR_WIDTH, SCR_HEIGHT, cameraPos, cameraFront, cameraUp, dis_scale);
}
int n = 0;
__host__ void display ( float *rho, float*ux, float *uy, float *uz,
                        float *rho_gpu, float *ux_gpu, float*uy_gpu, float* uz_gpu,
                        int NX, int NY, int NZ, 
                        ParticleSystem &fluid, glm::f32vec3 mod_scale, glm::f32vec3 dis_scale,
                        GLFWwindow** window, Shader& shader, Model &model, Geometry &fluidDomain, 
                        int num_mesh, Vertex **nodeLists, 
                        int *vertex_size_per_mesh,
                        hipStream_t *streams)
{
    glClearColor(0.45f, 0.85f, 0.15f, 0.05f);
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    draw_model( *window, shader, model, dis_scale, 
                num_mesh, nodeLists, vertex_size_per_mesh, streams);

    fluidDomain.draw_geometry(SCR_WIDTH, SCR_HEIGHT, cameraPos, cameraFront, cameraUp);
    
    draw_fluid(rho, ux, uy,uz,
               rho_gpu, ux_gpu, uy_gpu, uz_gpu,
               NX, NY, NZ, 
               fluid, mod_scale, dis_scale);

    // n++;
    // if((n%10000) == 0)
    // {
    //     std::ofstream o;
    //     char filename[128];
    //     char format[32];
    //     sprintf(filename,"00_%s_%d.csv","output",n);
    //     o.open(filename);
    //     for(int i=0;i<NZ;i++)
    //     {
    //         for(int j=0;j<NY;j++)
    //         {
    //             //setprecision(5);
    //             for(int k = 0;k<NX;k++)
    //                 o<<rho[i*(NX*NY) + NX*j + k]<<","<<ux[i*(NX*NY) + NX*j + k]<<","<<uy[i*(NX*NY) + NX*j + k]<<","<<uz[i*(NX*NY) + NX*j + k]<<","<<i<<","<<j<<","<<k<<std::endl;
    //         }
    //     }
    //     o<<std::endl;
    //     o.close();
    // }
    processInput(*window);
    glfwPollEvents();
    glfwSwapBuffers(*window);
}